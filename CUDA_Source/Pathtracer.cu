#include "hip/hip_runtime.h"
#include "cudart/hip/hip_vector_types.h"
#include "cudart/cuda_math.h"

#include "Common.h"

#define INFINITY ((float)(1e+300 * 1e+300))

__device__ __constant__ int screen_width;
__device__ __constant__ int screen_pitch;
__device__ __constant__ int screen_height;

__device__ __constant__ Config config;

#include "Util.h"
#include "Material.h"
#include "Sky.h"
#include "Sampling.h"
#include "RayCone.h"

// Frame Buffers
__device__ __constant__ float4 * frame_buffer_albedo;
__device__ __constant__ float4 * frame_buffer_direct;
__device__ __constant__ float4 * frame_buffer_indirect;

// Final Frame Buffer, shared with OpenGL
__device__ __constant__ Surface<float4> accumulator;

#include "Raytracing/BVH.h"
#include "Raytracing/QBVH.h"
#include "Raytracing/CWBVH.h"

#include "SVGF/SVGF.h"
#include "SVGF/TAA.h"

struct Camera {
	float3 position;
	float3 bottom_left_corner;
	float3 x_axis;
	float3 y_axis;
	float  pixel_spread_angle;
	float  aperture_radius;
	float  focal_distance;
} __device__ __constant__ camera;

__device__ PixelQuery pixel_query = { INVALID, INVALID, INVALID, INVALID };

extern "C" __global__ void kernel_generate(int sample_index, int pixel_offset, int pixel_count) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= pixel_count) return;

	int index_offset = index + pixel_offset;
	int x = index_offset % screen_width;
	int y = index_offset / screen_width;

	int pixel_index = x + y * screen_pitch;
	ASSERT(pixel_index < screen_pitch * screen_height, "Pixel should fit inside the buffer");

	float2 rand_filter   = random<SampleDimension::FILTER>  (pixel_index, 0, sample_index);
	float2 rand_aperture = random<SampleDimension::APERTURE>(pixel_index, 0, sample_index);

	float2 jitter;

	if (config.enable_svgf) {
		jitter.x = taa_halton_x[sample_index & (TAA_HALTON_NUM_SAMPLES-1)];
		jitter.y = taa_halton_y[sample_index & (TAA_HALTON_NUM_SAMPLES-1)];
	} else {
		switch (config.reconstruction_filter) {
			case ReconstructionFilter::BOX: {
				jitter = rand_filter;
				break;
			}
			case ReconstructionFilter::TENT: {
				jitter.x = sample_tent(rand_filter.x);
				jitter.y = sample_tent(rand_filter.y);
				break;
			}
			case ReconstructionFilter::GAUSSIAN: {
				float2 gaussians = sample_gaussian(rand_filter.x, rand_filter.y);
				jitter.x = 0.5f + 0.5f * gaussians.x;
				jitter.y = 0.5f + 0.5f * gaussians.y;
				break;
			}
		}
	}

	float x_jittered = float(x) + jitter.x;
	float y_jittered = float(y) + jitter.y;

	float3 focal_point = camera.focal_distance * normalize(camera.bottom_left_corner + x_jittered * camera.x_axis + y_jittered * camera.y_axis);
	float2 lens_point  = camera.aperture_radius * sample_disk(rand_aperture.x, rand_aperture.y);

	float3 offset = camera.x_axis * lens_point.x + camera.y_axis * lens_point.y;
	float3 direction = normalize(focal_point - offset);

	// Create primary Ray that starts at the Camera's position and goes through the current pixel
	ray_buffer_trace.origin   .set(index, camera.position + offset);
	ray_buffer_trace.direction.set(index, direction);

	ray_buffer_trace.pixel_index_and_mis_eligable[index] = pixel_index | (false << 31);
}

extern "C" __global__ void kernel_trace_bvh(int bounce) {
	bvh_trace(buffer_sizes.trace[bounce], &buffer_sizes.rays_retired[bounce]);
}

extern "C" __global__ void kernel_trace_qbvh(int bounce) {
	qbvh_trace(buffer_sizes.trace[bounce], &buffer_sizes.rays_retired[bounce]);
}

extern "C" __global__ void kernel_trace_cwbvh(int bounce) {
	cwbvh_trace(buffer_sizes.trace[bounce], &buffer_sizes.rays_retired[bounce]);
}

extern "C" __global__ void kernel_trace_shadow_bvh(int bounce) {
	bvh_trace_shadow(buffer_sizes.shadow[bounce], &buffer_sizes.rays_retired_shadow[bounce], bounce);
}

extern "C" __global__ void kernel_trace_shadow_qbvh(int bounce) {
	qbvh_trace_shadow(buffer_sizes.shadow[bounce], &buffer_sizes.rays_retired_shadow[bounce], bounce);
}

extern "C" __global__ void kernel_trace_shadow_cwbvh(int bounce) {
	cwbvh_trace_shadow(buffer_sizes.shadow[bounce], &buffer_sizes.rays_retired_shadow[bounce], bounce);
}

extern "C" __global__ void kernel_sort(int bounce, int sample_index) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.trace[bounce]) return;

	float3 ray_origin    = ray_buffer_trace.origin   .get(index);
	float3 ray_direction = ray_buffer_trace.direction.get(index);

	RayHit hit = ray_buffer_trace.hits.get(index);

	unsigned ray_pixel_index_and_mis_eligable = ray_buffer_trace.pixel_index_and_mis_eligable[index];
	int      ray_pixel_index = ray_pixel_index_and_mis_eligable & ~(0b11 << 31);

	int x = ray_pixel_index % screen_pitch;
	int y = ray_pixel_index / screen_pitch;

	bool mis_eligable = ray_pixel_index_and_mis_eligable >> 31;

	float3 ray_throughput;
	if (bounce == 0) {
		ray_throughput = make_float3(1.0f); // Throughput is known to be (1,1,1) still, skip the global memory load
	} else {
		ray_throughput = ray_buffer_trace.throughput.get(index);
	}

	// If we didn't hit anything, sample the Sky
	if (hit.triangle_id == INVALID) {
		float3 illumination = ray_throughput * sample_sky(ray_direction);

		if (bounce == 0) {
			if (config.enable_albedo || config.enable_svgf) {
				frame_buffer_albedo[ray_pixel_index] = make_float4(1.0f);
			}
			frame_buffer_direct[ray_pixel_index] = make_float4(illumination);
		} else if (bounce == 1) {
			frame_buffer_direct[ray_pixel_index] += make_float4(illumination);
		} else {
			frame_buffer_indirect[ray_pixel_index] += make_float4(illumination);
		}

		return;
	}

	// Get the Material of the Mesh we hit
	int material_id = mesh_get_material_id(hit.mesh_id);
	MaterialType material_type = material_get_type(material_id);

	if (bounce == 0 && pixel_query.pixel_index == ray_pixel_index) {
		pixel_query.mesh_id     = hit.mesh_id;
		pixel_query.triangle_id = hit.triangle_id;
		pixel_query.material_id = material_id;
	}

	if (material_type == MaterialType::LIGHT) {
		// Obtain the Light's position and normal
		TrianglePosNor light = triangle_get_positions_and_normals(hit.triangle_id);

		float3 light_point;
		float3 light_normal;
		triangle_barycentric(light, hit.u, hit.v, light_point, light_normal);

		float3 light_point_prev = light_point;

		// Transform into world space
		Matrix3x4 world = mesh_get_transform(hit.mesh_id);
		matrix3x4_transform_position (world, light_point);
		matrix3x4_transform_direction(world, light_normal);

		light_normal = normalize(light_normal);

		if (bounce == 0 && config.enable_svgf) {
			Matrix3x4 world_prev = mesh_get_transform_prev(hit.mesh_id);
			matrix3x4_transform_position(world_prev, light_point_prev);

			svgf_set_gbuffers(x, y, hit, light_point, light_normal, light_point_prev);
		}

		MaterialLight material_light = material_as_light(material_id);

		bool should_count_light_contribution = config.enable_next_event_estimation ? !mis_eligable : true;
		if (should_count_light_contribution) {
			float3 illumination = ray_throughput * material_light.emission;

			if (bounce == 0) {
				if (config.enable_albedo || config.enable_svgf) {
					frame_buffer_albedo[ray_pixel_index] = make_float4(1.0f);
				}
				frame_buffer_direct[ray_pixel_index] = make_float4(material_light.emission);
			} else if (bounce == 1) {
				frame_buffer_direct[ray_pixel_index] += make_float4(illumination);
			} else {
				frame_buffer_indirect[ray_pixel_index] += make_float4(illumination);
			}

			return;
		}

		if (config.enable_multiple_importance_sampling) {
			float3 to_light = light_point - ray_origin;
			float distance_to_light_squared = dot(to_light, to_light);
			float distance_to_light         = sqrtf(distance_to_light_squared);

			to_light /= distance_to_light; // Normalize

			float cos_theta_light = fabsf(dot(to_light, light_normal));

			float brdf_pdf = ray_buffer_trace.last_pdf[index];
			
			float light_power = luminance(material_light.emission.x, material_light.emission.y, material_light.emission.z);
			float light_pdf   = light_power * distance_to_light_squared / (cos_theta_light * lights_total_power);

			float weight = power_heuristic(brdf_pdf, light_pdf);
			float3 illumination = ray_throughput * material_light.emission * weight;

			assert(bounce != 0);
			if (bounce == 1) {
				frame_buffer_direct[ray_pixel_index] += make_float4(illumination);
			} else {
				frame_buffer_indirect[ray_pixel_index] += make_float4(illumination);
			}
		}

		return;
	}

	// Russian Roulette
	if (config.enable_russian_roulette && bounce > 0) {
		// Throughput does not include albedo so it doesn't need to be demodulated by SVGF (causing precision issues)
		// This deteriorates Russian Roulette performance, so albedo is included here
		float3 throughput_with_albedo = ray_throughput * make_float3(frame_buffer_albedo[ray_pixel_index]);

		float survival_probability  = saturate(vmax_max(throughput_with_albedo.x, throughput_with_albedo.y, throughput_with_albedo.z));
		float rand_russian_roulette = random<SampleDimension::RUSSIAN_ROULETTE>(ray_pixel_index, bounce, sample_index).x;

		if (rand_russian_roulette > survival_probability) {
			return;
		}

		ray_throughput /= survival_probability;
	}

	switch (material_type) {
		case MaterialType::DIFFUSE: {
			int index_out = atomic_agg_inc(&buffer_sizes.diffuse[bounce]);

			ray_buffer_shade_diffuse.direction.set(index_out, ray_direction);

			if (bounce > 0 && config.enable_mipmapping) ray_buffer_shade_diffuse.cone[index_out] = ray_buffer_trace.cone[index];

			ray_buffer_shade_diffuse.hits.set(index_out, hit);

			ray_buffer_shade_diffuse.pixel_index[index_out] = ray_pixel_index;
			if (bounce > 0) ray_buffer_shade_diffuse.throughput.set(index_out, ray_throughput);

			break;
		}

		case MaterialType::DIELECTRIC: {
			int index_out = atomic_agg_inc(&buffer_sizes.dielectric[bounce]);

			ray_buffer_shade_dielectric_and_glossy.direction.set(index_out, ray_direction);

			if (bounce > 0 && config.enable_mipmapping) ray_buffer_shade_dielectric_and_glossy.cone[index_out] = ray_buffer_trace.cone[index];

			ray_buffer_shade_dielectric_and_glossy.hits.set(index_out, hit);

			ray_buffer_shade_dielectric_and_glossy.pixel_index[index_out] = ray_pixel_index;
			if (bounce > 0) ray_buffer_shade_dielectric_and_glossy.throughput.set(index_out, ray_throughput);

			break;
		}

		case MaterialType::GLOSSY: {
			// Glossy Material buffer is shared with Dielectric Material buffer but grows in the opposite direction
			int index_out = (BATCH_SIZE - 1) - atomic_agg_inc(&buffer_sizes.glossy[bounce]);

			ray_buffer_shade_dielectric_and_glossy.direction.set(index_out, ray_direction);

			if (bounce > 0 && config.enable_mipmapping) ray_buffer_shade_dielectric_and_glossy.cone[index_out] = ray_buffer_trace.cone[index];

			ray_buffer_shade_dielectric_and_glossy.hits.set(index_out, hit);

			ray_buffer_shade_dielectric_and_glossy.pixel_index[index_out] = ray_pixel_index;
			if (bounce > 0) ray_buffer_shade_dielectric_and_glossy.throughput.set(index_out, ray_throughput);

			break;
		}
	}
}

__device__ inline float3 sample_albedo(
	int                       bounce,
	const float3            & material_diffuse,
	int                       material_texture_id,
	const RayHit            & hit,
	const TrianglePosNorTex & hit_triangle,
	const float3            & hit_point_local,
	const float3            & hit_normal,
	const float2            & hit_tex_coord,
	const float3            & ray_direction,
	const float2            * cone_buffer,
	int                       cone_buffer_index,
	float                   & cone_angle,
	float                   & cone_width
) {
	float3 albedo;

	float3 geometric_normal = cross(hit_triangle.position_edge_1, hit_triangle.position_edge_2);
	float  triangle_area_inv = 1.0f / length(geometric_normal);
	geometric_normal *= triangle_area_inv; // Normalize

	float mesh_scale = mesh_get_scale(hit.mesh_id);

	if (bounce == 0) {
		cone_angle = camera.pixel_spread_angle;
		cone_width = cone_angle * hit.t;

		float3 ellipse_axis_1, ellipse_axis_2; ray_cone_get_ellipse_axes(ray_direction, hit_normal, cone_width, ellipse_axis_1, ellipse_axis_2);

		float2 gradient_1, gradient_2; ray_cone_get_texture_gradients(
			mesh_scale,
			geometric_normal,
			triangle_area_inv,
			hit_triangle.position_0,  hit_triangle.position_edge_1,  hit_triangle.position_edge_2,
			hit_triangle.tex_coord_0, hit_triangle.tex_coord_edge_1, hit_triangle.tex_coord_edge_2,
			hit_point_local, hit_tex_coord,
			ellipse_axis_1, ellipse_axis_2,
			gradient_1, gradient_2
		);

		// Anisotropic sampling
		albedo = material_get_albedo(material_diffuse, material_texture_id, hit_tex_coord.x, hit_tex_coord.y, gradient_1, gradient_2);
	} else {
		float2 cone = cone_buffer[cone_buffer_index];
		cone_angle = cone.x;
		cone_width = cone.y + cone_angle * hit.t;

		float2 tex_size = texture_get_size(material_texture_id);

		float lod_triangle = sqrtf(tex_size.x * tex_size.y * triangle_get_lod(mesh_scale, triangle_area_inv, hit_triangle.tex_coord_edge_1, hit_triangle.tex_coord_edge_2));
		float lod_ray_cone = ray_cone_get_lod(ray_direction, hit_normal, cone_width);
		float lod = log2f(lod_triangle * lod_ray_cone);

		// Trilinear sampling
		albedo = material_get_albedo(material_diffuse, material_texture_id, hit_tex_coord.x, hit_tex_coord.y, lod);
	}

	float curvature = triangle_get_curvature(
		hit_triangle.position_edge_1,
		hit_triangle.position_edge_2,
		hit_triangle.normal_edge_1,
		hit_triangle.normal_edge_2
	) / mesh_scale;

	cone_angle += -2.0f * curvature * fabsf(cone_width / dot(hit_normal, ray_direction)); // Eq. 5 (Akenine-Möller 2021)

	return albedo;
}

template<typename BRDFEvaluator>
__device__ inline void nee_sample(
	int pixel_index,
	int bounce,
	int sample_index,
	const float3 & hit_point,
	const float3 & hit_normal,
	const float3 & throughput,
	BRDFEvaluator brdf_evaluator
) {
	float2 rand_light    = random<SampleDimension::NEE_LIGHT>   (pixel_index, bounce, sample_index);
	float2 rand_triangle = random<SampleDimension::NEE_TRIANGLE>(pixel_index, bounce, sample_index);

	// Pick random Light
	int light_mesh_id;
	int light_triangle_id = sample_light(rand_light.x, rand_light.y, light_mesh_id);

	// Pick random point on the Light
	float2 light_uv = sample_triangle(rand_triangle.x, rand_triangle.y);

	// Obtain the Light's position and normal
	TrianglePosNor light = triangle_get_positions_and_normals(light_triangle_id);

	float3 light_point;
	float3 light_normal;
	triangle_barycentric(light, light_uv.x, light_uv.y, light_point, light_normal);

	// Transform into world space
	Matrix3x4 light_world = mesh_get_transform(light_mesh_id);
	matrix3x4_transform_position (light_world, light_point);
	matrix3x4_transform_direction(light_world, light_normal);

	light_normal = normalize(light_normal);

	float3 to_light = light_point - hit_point;
	float distance_to_light_squared = dot(to_light, to_light);
	float distance_to_light         = sqrtf(distance_to_light_squared);

	// Normalize the vector to the light
	to_light /= distance_to_light;

	float cos_theta_light = fabsf(dot(to_light, light_normal));
	float cos_theta_hit   = dot(to_light, hit_normal);

	if (cos_theta_hit <= 0.0f) return; // No light transport possible

	int light_material_id = mesh_get_material_id(light_mesh_id);
	MaterialLight material_light = material_as_light(light_material_id);

	float  brdf_pdf;
	float3 brdf = brdf_evaluator(to_light, brdf_pdf);
	
	float light_power = luminance(material_light.emission.x, material_light.emission.y, material_light.emission.z);
	float light_pdf = light_power * distance_to_light_squared / (cos_theta_light * lights_total_power);

	float weight;
	if (config.enable_multiple_importance_sampling) {
		weight = power_heuristic(light_pdf, brdf_pdf);
	} else {
		weight = 1.0f;
	}

	float3 illumination = throughput * brdf * material_light.emission * cos_theta_hit * weight / light_pdf;

	int shadow_ray_index = atomic_agg_inc(&buffer_sizes.shadow[bounce]);

	ray_buffer_shadow.ray_origin   .set(shadow_ray_index, ray_origin_epsilon_offset(hit_point, hit_normal));
	ray_buffer_shadow.ray_direction.set(shadow_ray_index, to_light);

	ray_buffer_shadow.max_distance[shadow_ray_index] = distance_to_light - EPSILON;

	ray_buffer_shadow.illumination_and_pixel_index[shadow_ray_index] = make_float4(
		illumination.x,
		illumination.y,
		illumination.z,
		__int_as_float(pixel_index)
	);
}

extern "C" __global__ void kernel_shade_diffuse(int bounce, int sample_index) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.diffuse[bounce]) return;

	float3 ray_direction = ray_buffer_shade_diffuse.direction.get(index);
	RayHit hit           = ray_buffer_shade_diffuse.hits     .get(index);

	int ray_pixel_index = ray_buffer_shade_diffuse.pixel_index[index];
	int x = ray_pixel_index % screen_pitch;
	int y = ray_pixel_index / screen_pitch;

	float3 ray_throughput;
	if (bounce == 0) {
		ray_throughput = make_float3(1.0f); // Throughput is known to be (1,1,1) still, skip the global memory load
	} else {
		ray_throughput = ray_buffer_shade_diffuse.throughput.get(index);
	}

	int material_id = mesh_get_material_id(hit.mesh_id);
	MaterialDiffuse material = material_as_diffuse(material_id);

	// Obtain hit Triangle position, normal, and texture coordinates
	TrianglePosNorTex hit_triangle = triangle_get_positions_normals_and_tex_coords(hit.triangle_id);

	float3 hit_point;
	float3 hit_normal;
	float2 hit_tex_coord;
	triangle_barycentric(hit_triangle, hit.u, hit.v, hit_point, hit_normal, hit_tex_coord);

	float3 hit_point_local = hit_point; // Keep copy of the untransformed hit point in local space

	// Transform into world space
	Matrix3x4 world = mesh_get_transform(hit.mesh_id);
	matrix3x4_transform_position (world, hit_point);
	matrix3x4_transform_direction(world, hit_normal);

	hit_normal = normalize(hit_normal);
	if (dot(ray_direction, hit_normal) > 0.0f) hit_normal = -hit_normal;

	// Sample albedo
	float cone_angle;
	float cone_width;
	float3 albedo;
	if (config.enable_mipmapping) {
		albedo = sample_albedo(
			bounce,
			material.diffuse,
			material.texture_id,
			hit,
			hit_triangle,
			hit_point_local,
			hit_normal,
			hit_tex_coord,
			ray_direction,
			ray_buffer_shade_diffuse.cone,
			index,
			cone_angle, cone_width
		);
	} else {
		albedo = material_get_albedo(material.diffuse, material.texture_id, hit_tex_coord.x, hit_tex_coord.y);
	}

	float3 throughput = ray_throughput;

	if (bounce > 0) {
		throughput *= albedo;
	} else if (config.enable_albedo || config.enable_svgf) {
		frame_buffer_albedo[ray_pixel_index] = make_float4(albedo);
	}

	if (bounce == 0 && config.enable_svgf) {
		float3 hit_point_prev = hit_point_local;

		Matrix3x4 world_prev = mesh_get_transform_prev(hit.mesh_id);
		matrix3x4_transform_position(world_prev, hit_point_prev);

		svgf_set_gbuffers(x, y, hit, hit_point, hit_normal, hit_point_prev);
	}

	if (config.enable_next_event_estimation && lights_total_power > 0.0f) {
		nee_sample(ray_pixel_index, bounce, sample_index, hit_point, hit_normal, throughput, [&](const float3 & to_light, float & pdf) {
			pdf = dot(to_light, hit_normal) * ONE_OVER_PI;
			return make_float3(ONE_OVER_PI);
		});
	}

	if (bounce == config.num_bounces - 1) return;

	int index_out = atomic_agg_inc(&buffer_sizes.trace[bounce + 1]);

	float3 tangent, binormal; orthonormal_basis(hit_normal, tangent, binormal);

	float2 rand_brdf = random<SampleDimension::BRDF>(ray_pixel_index, bounce, sample_index);
	float3 direction_local = sample_cosine_weighted_direction(rand_brdf.x, rand_brdf.y);
	float3 direction_world = local_to_world(direction_local, tangent, binormal, hit_normal);

	ray_buffer_trace.origin   .set(index_out, ray_origin_epsilon_offset(hit_point, hit_normal));
	ray_buffer_trace.direction.set(index_out, direction_world);

	if (config.enable_mipmapping) {
		ray_buffer_trace.cone[index_out] = make_float2(cone_angle, cone_width);
	}
	
	ray_buffer_trace.pixel_index_and_mis_eligable[index_out] = ray_pixel_index | (true << 31);
	ray_buffer_trace.throughput.set(index_out, throughput);

	ray_buffer_trace.last_pdf[index_out] = fabsf(dot(direction_world, hit_normal)) * ONE_OVER_PI;
}

extern "C" __global__ void kernel_shade_dielectric(int bounce, int sample_index) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.dielectric[bounce] || bounce == config.num_bounces - 1) return;

	float3 ray_direction = ray_buffer_shade_dielectric_and_glossy.direction.get(index);
	RayHit hit           = ray_buffer_shade_dielectric_and_glossy.hits     .get(index);

	int ray_pixel_index = ray_buffer_shade_dielectric_and_glossy.pixel_index[index];

	float3 ray_throughput;
	if (bounce == 0) {
		ray_throughput = make_float3(1.0f); // Throughput is known to be (1,1,1) still, skip the global memory load
	} else {
		ray_throughput = ray_buffer_shade_dielectric_and_glossy.throughput.get(index);
	}

	ASSERT(hit.triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	int material_id = mesh_get_material_id(hit.mesh_id);
	MaterialDielectric material = material_as_dielectric(material_id);

	// Obtain hit Triangle position, normal, and texture coordinates
	TrianglePosNor hit_triangle = triangle_get_positions_and_normals(hit.triangle_id);

	float3 hit_point;
	float3 hit_normal;
	triangle_barycentric(hit_triangle, hit.u, hit.v, hit_point, hit_normal);

	// Transform into world space
	Matrix3x4 world = mesh_get_transform(hit.mesh_id);
	matrix3x4_transform_position (world, hit_point);
	matrix3x4_transform_direction(world, hit_normal);

	hit_normal = normalize(hit_normal);

	// Calculate proper facing normal and determine index of refraction
	float cos_theta = dot(ray_direction, hit_normal);
	float eta;
	if (cos_theta < 0.0f) {
		// Entering material
		eta = 1.0f / material.index_of_refraction;

		hit_normal =  hit_normal;
		cos_theta  = -cos_theta;
	} else {
		// Leaving material
		eta = material.index_of_refraction;

		hit_normal = -hit_normal;
		cos_theta  =  cos_theta;

		// Lambert-Beer Law
		// NOTE: does not take into account e.g. nested dielectrics or diffuse inside dielectric!
		ray_throughput.x *= expf(material.negative_absorption.x * hit.t);
		ray_throughput.y *= expf(material.negative_absorption.y * hit.t);
		ray_throughput.z *= expf(material.negative_absorption.z * hit.t);
	}

	float k = 1.0f - eta*eta * (1.0f - cos_theta*cos_theta);

	float3 ray_direction_reflected = reflect(ray_direction, hit_normal);
	float3 direction_out;
	float3 origin_out;

	if (k < 0.0f) { // Total Internal Reflection
		direction_out = ray_direction_reflected;
	} else {
		float3 ray_direction_refracted = normalize(eta * ray_direction + (eta * cos_theta - sqrtf(k)) * hit_normal);

		float fresnel      = fresnel_dielectric(cos_theta, -dot(ray_direction_refracted, hit_normal), eta);
		float rand_fresnel = random<SampleDimension::BRDF>(ray_pixel_index, bounce, sample_index).x;

		if (rand_fresnel < fresnel) {
			direction_out = ray_direction_reflected;
			origin_out    = ray_origin_epsilon_offset(hit_point, hit_normal);
		} else {
			direction_out = ray_direction_refracted;
			origin_out    = ray_origin_epsilon_offset(hit_point, -hit_normal);
		}
	}

	if (bounce == 0 && (config.enable_albedo || config.enable_svgf)) {
		frame_buffer_albedo[ray_pixel_index] = make_float4(1.0f);
	}

	int index_out = atomic_agg_inc(&buffer_sizes.trace[bounce + 1]);

	ray_buffer_trace.origin   .set(index_out, origin_out);
	ray_buffer_trace.direction.set(index_out, direction_out);

	if (config.enable_mipmapping) {
		float2 cone = ray_buffer_shade_dielectric_and_glossy.cone[index];
		float  cone_angle = cone.x;
		float  cone_width = cone.y + cone_angle * hit.t;

		float mesh_scale = mesh_get_scale(hit.mesh_id);

		float curvature = triangle_get_curvature(
			hit_triangle.position_edge_1,
			hit_triangle.position_edge_2,
			hit_triangle.normal_edge_1,
			hit_triangle.normal_edge_2
		) / mesh_scale;

		cone_angle += -2.0f * curvature * fabsf(cone_width) / dot(hit_normal, ray_direction); // Eq. 5 (Akenine-Möller 2021)

		ray_buffer_trace.cone[index_out] = make_float2(cone_angle, cone_width);
	}
	
	ray_buffer_trace.pixel_index_and_mis_eligable[index_out] = ray_pixel_index | (false << 31);
	ray_buffer_trace.throughput.set(index_out, ray_throughput);
}

extern "C" __global__ void kernel_shade_glossy(int bounce, int sample_index) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.glossy[bounce]) return;

	index = (BATCH_SIZE - 1) - index;

	float3 ray_direction = ray_buffer_shade_dielectric_and_glossy.direction.get(index);

	RayHit hit = ray_buffer_shade_dielectric_and_glossy.hits.get(index);

	int ray_pixel_index = ray_buffer_shade_dielectric_and_glossy.pixel_index[index];
	int x = ray_pixel_index % screen_pitch;
	int y = ray_pixel_index / screen_pitch;

	float3 ray_throughput;
	if (bounce == 0) {
		ray_throughput = make_float3(1.0f);	// Throughput is known to be (1,1,1) still, skip the global memory load
	} else {
		ray_throughput = ray_buffer_shade_dielectric_and_glossy.throughput.get(index);
	}

	ASSERT(hit.triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	int material_id = mesh_get_material_id(hit.mesh_id);
	MaterialGlossy material = material_as_glossy(material_id);

	// Obtain hit Triangle position, normal, and texture coordinates
	TrianglePosNorTex hit_triangle = triangle_get_positions_normals_and_tex_coords(hit.triangle_id);

	float3 hit_point;
	float3 hit_normal;
	float2 hit_tex_coord;
	triangle_barycentric(hit_triangle, hit.u, hit.v, hit_point, hit_normal, hit_tex_coord);

	float3 hit_point_local = hit_point; // Keep copy of the untransformed hit point in local space

	// Transform into world space
	Matrix3x4 world = mesh_get_transform(hit.mesh_id);
	matrix3x4_transform_position (world, hit_point);
	matrix3x4_transform_direction(world, hit_normal);

	hit_normal = normalize(hit_normal);
	if (dot(ray_direction, hit_normal) > 0.0f) hit_normal = -hit_normal;

	// Sample albedo
	float cone_angle;
	float cone_width;
	float3 albedo;
	if (config.enable_mipmapping) {
		albedo = sample_albedo(
			bounce,
			material.diffuse,
			material.texture_id,
			hit,
			hit_triangle,
			hit_point_local,
			hit_normal,
			hit_tex_coord,
			ray_direction,
			ray_buffer_shade_dielectric_and_glossy.cone,
			index,
			cone_angle, cone_width
		);
	} else {
		albedo = material_get_albedo(material.diffuse, material.texture_id, hit_tex_coord.x, hit_tex_coord.y);
	}

	float3 throughput = ray_throughput;

	if (bounce > 0) {
		throughput *= albedo;
	} else if (config.enable_albedo || config.enable_svgf) {
		frame_buffer_albedo[ray_pixel_index] = make_float4(albedo);
	}

	if (bounce == 0 && config.enable_svgf) {
		float3 hit_point_prev = hit_point_local;

		Matrix3x4 world_prev = mesh_get_transform_prev(hit.mesh_id);
		matrix3x4_transform_position(world_prev, hit_point_prev);

		svgf_set_gbuffers(x, y, hit, hit_point, hit_normal, hit_point_prev);
	}

	// Construct orthonormal basis
	float3 hit_tangent, hit_binormal;
	orthonormal_basis(hit_normal, hit_tangent, hit_binormal);

	float3 omega_i = world_to_local(-ray_direction, hit_tangent, hit_binormal, hit_normal);

	if (config.enable_next_event_estimation && lights_total_power > 0.0f && material.roughness >= ROUGHNESS_CUTOFF) {
		nee_sample(ray_pixel_index, bounce, sample_index, hit_point, hit_normal, throughput, [&](const float3 & to_light, float & pdf) {
			float3 omega_o = world_to_local(to_light, hit_tangent, hit_binormal, hit_normal);
			return ggx_eval(material, omega_o, omega_i, pdf);
		});
	}

	if (bounce == config.num_bounces - 1) return;

	// Importance sample distribution of normals
	float2 rand_brdf = random<SampleDimension::BRDF>(ray_pixel_index, bounce, sample_index);

	float3 micro_normal_local = sample_ggx_distribution_of_normals(omega_i, material.roughness, material.roughness, rand_brdf.x, rand_brdf.y);
	float3 omega_o = reflect(-omega_i, micro_normal_local);

	float pdf;
	throughput *= ggx_eval(material, omega_o, omega_i, pdf) * omega_o.z;
	throughput /= pdf;

	float3 direction_out = local_to_world(omega_o, hit_tangent, hit_binormal, hit_normal);
	
	int index_out = atomic_agg_inc(&buffer_sizes.trace[bounce + 1]);

	ray_buffer_trace.origin   .set(index_out, ray_origin_epsilon_offset(hit_point, hit_normal));
	ray_buffer_trace.direction.set(index_out, direction_out);

	if (config.enable_mipmapping) {
		ray_buffer_trace.cone[index_out] = make_float2(cone_angle, cone_width);
	}
	
	ray_buffer_trace.pixel_index_and_mis_eligable[index_out] = ray_pixel_index | ((material.roughness >= ROUGHNESS_CUTOFF) << 31);
	ray_buffer_trace.throughput.set(index_out, throughput);

	ray_buffer_trace.last_pdf[index_out] = pdf;
}

extern "C" __global__ void kernel_accumulate(float frames_accumulated) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= screen_width || y >= screen_height) return;

	int pixel_index = x + y * screen_pitch;

	float4 direct   = frame_buffer_direct  [pixel_index];
	float4 indirect = frame_buffer_indirect[pixel_index];

	float4 colour = direct + indirect;

	if (config.enable_albedo) {
		colour *= frame_buffer_albedo[pixel_index];
	}

	if (frames_accumulated > 0.0f) {
		float4 colour_prev = accumulator.get(x, y);

		colour = colour_prev + (colour - colour_prev) / frames_accumulated; // Online average
	}

	accumulator.set(x, y, colour);
}
