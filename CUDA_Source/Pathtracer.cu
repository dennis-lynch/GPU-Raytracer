#include "hip/hip_runtime.h"
#include "cudart/hip/hip_vector_types.h"
#include "cudart/cuda_math.h"

#include "Common.h"

#define INFINITY ((float)(1e+300 * 1e+300))

__device__ __constant__ int screen_width;
__device__ __constant__ int screen_pitch;
__device__ __constant__ int screen_height;

__device__ __constant__ Config config;

#include "Util.h"
#include "Material.h"
#include "Sky.h"
#include "RayCone.h"

// Frame Buffers
__device__ __constant__ float4 * frame_buffer_albedo;
__device__ __constant__ float4 * frame_buffer_direct;
__device__ __constant__ float4 * frame_buffer_indirect;

// Final Frame Buffer, shared with OpenGL
__device__ __constant__ Surface<float4> accumulator;

#include "Raytracing/BVH.h"
#include "Raytracing/QBVH.h"
#include "Raytracing/CWBVH.h"

#include "Sampling.h"

#include "SVGF/SVGF.h"
#include "SVGF/TAA.h"

struct Camera {
	float3 position;
	float3 bottom_left_corner;
	float3 x_axis;
	float3 y_axis;
	float  pixel_spread_angle;
	float  aperture_radius;
	float  focal_distance;
} __device__ __constant__ camera;

__device__ PixelQuery pixel_query = { INVALID, INVALID, INVALID, INVALID };

extern "C" __global__ void kernel_generate(int sample_index, int pixel_offset, int pixel_count) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= pixel_count) return;

	int index_offset = index + pixel_offset;
	int x = index_offset % screen_width;
	int y = index_offset / screen_width;

	int pixel_index = x + y * screen_pitch;
	ASSERT(pixel_index < screen_pitch * screen_height, "Pixel should fit inside the buffer");

	float2 rand_filter   = random<SampleDimension::FILTER>  (pixel_index, 0, sample_index);
	float2 rand_aperture = random<SampleDimension::APERTURE>(pixel_index, 0, sample_index);

	float2 jitter;

	if (config.enable_svgf) {
		jitter.x = taa_halton_x[sample_index & (TAA_HALTON_NUM_SAMPLES-1)];
		jitter.y = taa_halton_y[sample_index & (TAA_HALTON_NUM_SAMPLES-1)];
	} else {
		switch (config.reconstruction_filter) {
			case ReconstructionFilter::BOX: {
				jitter = rand_filter;
				break;
			}
			case ReconstructionFilter::TENT: {
				jitter.x = sample_tent(rand_filter.x);
				jitter.y = sample_tent(rand_filter.y);
				break;
			}
			case ReconstructionFilter::GAUSSIAN: {
				float2 gaussians = sample_gaussian(rand_filter.x, rand_filter.y);
				jitter.x = 0.5f + 0.5f * gaussians.x;
				jitter.y = 0.5f + 0.5f * gaussians.y;
				break;
			}
		}
	}

	float x_jittered = float(x) + jitter.x;
	float y_jittered = float(y) + jitter.y;

	float3 focal_point = camera.focal_distance * normalize(camera.bottom_left_corner + x_jittered * camera.x_axis + y_jittered * camera.y_axis);
	float2 lens_point  = camera.aperture_radius * sample_disk(rand_aperture.x, rand_aperture.y);

	float3 offset = camera.x_axis * lens_point.x + camera.y_axis * lens_point.y;
	float3 direction = normalize(focal_point - offset);

	// Create primary Ray that starts at the Camera's position and goes through the current pixel
	ray_buffer_trace.origin   .set(index, camera.position + offset);
	ray_buffer_trace.direction.set(index, direction);

	ray_buffer_trace.pixel_index_and_mis_eligable[index] = pixel_index | (false << 31);
}

extern "C" __global__ void kernel_trace_bvh(int bounce) {
	bvh_trace(buffer_sizes.trace[bounce], &buffer_sizes.rays_retired[bounce]);
}

extern "C" __global__ void kernel_trace_qbvh(int bounce) {
	qbvh_trace(buffer_sizes.trace[bounce], &buffer_sizes.rays_retired[bounce]);
}

extern "C" __global__ void kernel_trace_cwbvh(int bounce) {
	cwbvh_trace(buffer_sizes.trace[bounce], &buffer_sizes.rays_retired[bounce]);
}

extern "C" __global__ void kernel_trace_shadow_bvh(int bounce) {
	bvh_trace_shadow(buffer_sizes.shadow[bounce], &buffer_sizes.rays_retired_shadow[bounce], bounce);
}

extern "C" __global__ void kernel_trace_shadow_qbvh(int bounce) {
	qbvh_trace_shadow(buffer_sizes.shadow[bounce], &buffer_sizes.rays_retired_shadow[bounce], bounce);
}

extern "C" __global__ void kernel_trace_shadow_cwbvh(int bounce) {
	cwbvh_trace_shadow(buffer_sizes.shadow[bounce], &buffer_sizes.rays_retired_shadow[bounce], bounce);
}

extern "C" __global__ void kernel_sort(int bounce, int sample_index) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.trace[bounce]) return;

	float3 ray_direction = ray_buffer_trace.direction.get(index);

	RayHit hit = ray_buffer_trace.hits.get(index);

	unsigned ray_pixel_index_and_mis_eligable = ray_buffer_trace.pixel_index_and_mis_eligable[index];
	int      ray_pixel_index = ray_pixel_index_and_mis_eligable & ~(0b11 << 31);

	int x = ray_pixel_index % screen_pitch;
	int y = ray_pixel_index / screen_pitch;

	bool mis_eligable = ray_pixel_index_and_mis_eligable >> 31;

	float3 ray_throughput;
	if (bounce == 0) {
		ray_throughput = make_float3(1.0f); // Throughput is known to be (1,1,1) still, skip the global memory load
	} else {
		ray_throughput = ray_buffer_trace.throughput.get(index);
	}

	// If we didn't hit anything, sample the Sky
	if (hit.triangle_id == INVALID) {
		float3 illumination = ray_throughput * sample_sky(ray_direction);

		if (bounce == 0) {
			if (config.enable_albedo || config.enable_svgf) {
				frame_buffer_albedo[ray_pixel_index] = make_float4(1.0f);
			}
			frame_buffer_direct[ray_pixel_index] = make_float4(illumination);
		} else if (bounce == 1) {
			frame_buffer_direct[ray_pixel_index] += make_float4(illumination);
		} else {
			frame_buffer_indirect[ray_pixel_index] += make_float4(illumination);
		}

		return;
	}

	// Get the Material of the Mesh we hit
	int material_id = mesh_get_material_id(hit.mesh_id);
	MaterialType material_type = material_get_type(material_id);

	if (bounce == 0 && pixel_query.pixel_index == ray_pixel_index) {
		pixel_query.mesh_id     = hit.mesh_id;
		pixel_query.triangle_id = hit.triangle_id;
		pixel_query.material_id = material_id;
	}

	if (material_type == MaterialType::LIGHT) {
		// Obtain the Light's position and normal
		TrianglePosNor light = triangle_get_positions_and_normals(hit.triangle_id);

		float3 light_point;
		float3 light_normal;
		triangle_barycentric(light, hit.u, hit.v, light_point, light_normal);

		float3 light_point_prev = light_point;

		// Transform into world space
		Matrix3x4 world = mesh_get_transform(hit.mesh_id);
		matrix3x4_transform_position (world, light_point);
		matrix3x4_transform_direction(world, light_normal);

		light_normal = normalize(light_normal);

		if (bounce == 0 && config.enable_svgf) {
			Matrix3x4 world_prev = mesh_get_transform_prev(hit.mesh_id);
			matrix3x4_transform_position(world_prev, light_point_prev);

			svgf_set_gbuffers(x, y, hit, light_point, light_normal, light_point_prev);
		}

		MaterialLight material_light = material_as_light(material_id);

		bool should_count_light_contribution = config.enable_next_event_estimation ? !mis_eligable : true;
		if (should_count_light_contribution) {
			float3 illumination = ray_throughput * material_light.emission;

			if (bounce == 0) {
				if (config.enable_albedo || config.enable_svgf) {
					frame_buffer_albedo[ray_pixel_index] = make_float4(1.0f);
				}
				frame_buffer_direct[ray_pixel_index] = make_float4(material_light.emission);
			} else if (bounce == 1) {
				frame_buffer_direct[ray_pixel_index] += make_float4(illumination);
			} else {
				frame_buffer_indirect[ray_pixel_index] += make_float4(illumination);
			}

			return;
		}

		if (config.enable_multiple_importance_sampling) {
			float cos_theta_light = fabsf(dot(ray_direction, light_normal));
			float distance_to_light_squared = hit.t * hit.t;

			float brdf_pdf = ray_buffer_trace.last_pdf[index];

			float light_power = luminance(material_light.emission.x, material_light.emission.y, material_light.emission.z);
			float light_pdf   = light_power * distance_to_light_squared / (cos_theta_light * lights_total_weight);

			float mis_weight = power_heuristic(brdf_pdf, light_pdf);
			float3 illumination = ray_throughput * material_light.emission * mis_weight;

			assert(bounce != 0);
			if (bounce == 1) {
				frame_buffer_direct[ray_pixel_index] += make_float4(illumination);
			} else {
				frame_buffer_indirect[ray_pixel_index] += make_float4(illumination);
			}
		}

		return;
	}

	// If this is the last bounce and we haven't hit a light, terminate
	if (bounce == config.num_bounces - 1) return;

	// Russian Roulette
	if (config.enable_russian_roulette && bounce > 0) {
		// Throughput does not include albedo so it doesn't need to be demodulated by SVGF (causing precision issues)
		// This deteriorates Russian Roulette performance, so albedo is included here
		float3 throughput_with_albedo = ray_throughput * make_float3(frame_buffer_albedo[ray_pixel_index]);

		float survival_probability  = saturate(vmax_max(throughput_with_albedo.x, throughput_with_albedo.y, throughput_with_albedo.z));
		float rand_russian_roulette = random<SampleDimension::RUSSIAN_ROULETTE>(ray_pixel_index, bounce, sample_index).x;

		if (rand_russian_roulette > survival_probability) {
			return;
		}

		ray_throughput /= survival_probability;
	}

	switch (material_type) {
		case MaterialType::DIFFUSE: {
			int index_out = atomic_agg_inc(&buffer_sizes.diffuse[bounce]);

			ray_buffer_shade_diffuse_and_plastic.direction.set(index_out, ray_direction);

			if (bounce > 0 && config.enable_mipmapping) ray_buffer_shade_diffuse_and_plastic.cone[index_out] = ray_buffer_trace.cone[index];

			ray_buffer_shade_diffuse_and_plastic.hits.set(index_out, hit);

			ray_buffer_shade_diffuse_and_plastic.pixel_index[index_out] = ray_pixel_index;
			if (bounce > 0) ray_buffer_shade_diffuse_and_plastic.throughput.set(index_out, ray_throughput);

			break;
		}

		case MaterialType::PLASTIC: {
			// Plastic Material buffer is shared with Diffuse Material buffer but grows in the opposite direction
			int index_out = (BATCH_SIZE - 1) - atomic_agg_inc(&buffer_sizes.plastic[bounce]);

			ray_buffer_shade_diffuse_and_plastic.direction.set(index_out, ray_direction);

			if (bounce > 0 && config.enable_mipmapping) ray_buffer_shade_diffuse_and_plastic.cone[index_out] = ray_buffer_trace.cone[index];

			ray_buffer_shade_diffuse_and_plastic.hits.set(index_out, hit);

			ray_buffer_shade_diffuse_and_plastic.pixel_index[index_out] = ray_pixel_index;
			if (bounce > 0) ray_buffer_shade_diffuse_and_plastic.throughput.set(index_out, ray_throughput);

			break;
		}

		case MaterialType::DIELECTRIC: {
			int index_out = atomic_agg_inc(&buffer_sizes.dielectric[bounce]);

			ray_buffer_shade_dielectric_and_conductor.direction.set(index_out, ray_direction);

			if (bounce > 0 && config.enable_mipmapping) ray_buffer_shade_dielectric_and_conductor.cone[index_out] = ray_buffer_trace.cone[index];

			ray_buffer_shade_dielectric_and_conductor.hits.set(index_out, hit);

			ray_buffer_shade_dielectric_and_conductor.pixel_index[index_out] = ray_pixel_index;
			if (bounce > 0) ray_buffer_shade_dielectric_and_conductor.throughput.set(index_out, ray_throughput);

			break;
		}

		case MaterialType::CONDUCTOR: {
			// Conductor Material buffer is shared with Dielectric Material buffer but grows in the opposite direction
			int index_out = (BATCH_SIZE - 1) - atomic_agg_inc(&buffer_sizes.conductor[bounce]);

			ray_buffer_shade_dielectric_and_conductor.direction.set(index_out, ray_direction);

			if (bounce > 0 && config.enable_mipmapping) ray_buffer_shade_dielectric_and_conductor.cone[index_out] = ray_buffer_trace.cone[index];

			ray_buffer_shade_dielectric_and_conductor.hits.set(index_out, hit);

			ray_buffer_shade_dielectric_and_conductor.pixel_index[index_out] = ray_pixel_index;
			if (bounce > 0) ray_buffer_shade_dielectric_and_conductor.throughput.set(index_out, ray_throughput);

			break;
		}
	}
}

__device__ inline float3 sample_albedo(
	int                       bounce,
	const float3            & material_diffuse,
	int                       material_texture_id,
	const RayHit            & hit,
	const TrianglePosNorTex & hit_triangle,
	const float3            & hit_point_local,
	const float3            & hit_normal,
	const float2            & hit_tex_coord,
	const float3            & ray_direction,
	const float2            * cone_buffer,
	int                       cone_buffer_index,
	float                   & cone_angle,
	float                   & cone_width
) {
	float3 albedo;

	float3 geometric_normal = cross(hit_triangle.position_edge_1, hit_triangle.position_edge_2);
	float  triangle_area_inv = 1.0f / length(geometric_normal);
	geometric_normal *= triangle_area_inv; // Normalize

	float mesh_scale = mesh_get_scale(hit.mesh_id);

	if (bounce == 0) {
		cone_angle = camera.pixel_spread_angle;
		cone_width = cone_angle * hit.t;

		float3 ellipse_axis_1, ellipse_axis_2; ray_cone_get_ellipse_axes(ray_direction, hit_normal, cone_width, ellipse_axis_1, ellipse_axis_2);

		float2 gradient_1, gradient_2; ray_cone_get_texture_gradients(
			mesh_scale,
			geometric_normal,
			triangle_area_inv,
			hit_triangle.position_0,  hit_triangle.position_edge_1,  hit_triangle.position_edge_2,
			hit_triangle.tex_coord_0, hit_triangle.tex_coord_edge_1, hit_triangle.tex_coord_edge_2,
			hit_point_local, hit_tex_coord,
			ellipse_axis_1, ellipse_axis_2,
			gradient_1, gradient_2
		);

		// Anisotropic sampling
		albedo = material_get_albedo(material_diffuse, material_texture_id, hit_tex_coord.x, hit_tex_coord.y, gradient_1, gradient_2);
	} else {
		float2 cone = cone_buffer[cone_buffer_index];
		cone_angle = cone.x;
		cone_width = cone.y + cone_angle * hit.t;

		float2 tex_size = texture_get_size(material_texture_id);

		float lod_triangle = sqrtf(tex_size.x * tex_size.y * triangle_get_lod(mesh_scale, triangle_area_inv, hit_triangle.tex_coord_edge_1, hit_triangle.tex_coord_edge_2));
		float lod_ray_cone = ray_cone_get_lod(ray_direction, hit_normal, cone_width);
		float lod = log2f(lod_triangle * lod_ray_cone);

		// Trilinear sampling
		albedo = material_get_albedo(material_diffuse, material_texture_id, hit_tex_coord.x, hit_tex_coord.y, lod);
	}

	float curvature = triangle_get_curvature(
		hit_triangle.position_edge_1,
		hit_triangle.position_edge_2,
		hit_triangle.normal_edge_1,
		hit_triangle.normal_edge_2
	) / mesh_scale;

	cone_angle += -2.0f * curvature * fabsf(cone_width / dot(hit_normal, ray_direction)); // Eq. 5 (Akenine-Möller 2021)

	return albedo;
}

template<typename BRDFEvaluator>
__device__ inline void nee_sample(
	int pixel_index,
	int bounce,
	int sample_index,
	const float3 & hit_point,
	const float3 & hit_normal,
	const float3 & throughput,
	BRDFEvaluator brdf_evaluator
) {
	float2 rand_light    = random<SampleDimension::NEE_LIGHT>   (pixel_index, bounce, sample_index);
	float2 rand_triangle = random<SampleDimension::NEE_TRIANGLE>(pixel_index, bounce, sample_index);

	// Pick random Light
	int light_mesh_id;
	int light_triangle_id = sample_light(rand_light.x, rand_light.y, light_mesh_id);

	// Pick random point on the Light
	float2 light_uv = sample_triangle(rand_triangle.x, rand_triangle.y);

	// Obtain the Light's position and normal
	TrianglePosNor light = triangle_get_positions_and_normals(light_triangle_id);

	float3 light_point;
	float3 light_normal;
	triangle_barycentric(light, light_uv.x, light_uv.y, light_point, light_normal);

	// Transform into world space
	Matrix3x4 light_world = mesh_get_transform(light_mesh_id);
	matrix3x4_transform_position (light_world, light_point);
	matrix3x4_transform_direction(light_world, light_normal);

	light_normal = normalize(light_normal);

	float3 to_light = light_point - hit_point;
	float distance_to_light_squared = dot(to_light, to_light);
	float distance_to_light         = sqrtf(distance_to_light_squared);

	// Normalize the vector to the light
	to_light /= distance_to_light;

	float cos_theta_light = fabsf(dot(to_light, light_normal));
	float cos_theta_hit = dot(to_light, hit_normal);

	int light_material_id = mesh_get_material_id(light_mesh_id);
	MaterialLight material_light = material_as_light(light_material_id);

	float3 brdf;
	float  brdf_pdf;
	bool valid = brdf_evaluator(to_light, cos_theta_hit, brdf, brdf_pdf);

	if (!valid) return;
	assert(brdf_pdf != 0.0f);

	float light_power = luminance(material_light.emission.x, material_light.emission.y, material_light.emission.z);
	float light_pdf   = light_power * distance_to_light_squared / (cos_theta_light * lights_total_weight);

	float mis_weight;
	if (config.enable_multiple_importance_sampling) {
		mis_weight = power_heuristic(light_pdf, brdf_pdf);
	} else {
		mis_weight = 1.0f;
	}

	float3 illumination = throughput * brdf * material_light.emission * mis_weight / light_pdf;

	int shadow_ray_index = atomic_agg_inc(&buffer_sizes.shadow[bounce]);

	ray_buffer_shadow.ray_origin   .set(shadow_ray_index, ray_origin_epsilon_offset(hit_point, to_light, hit_normal));
	ray_buffer_shadow.ray_direction.set(shadow_ray_index, to_light);

	ray_buffer_shadow.max_distance[shadow_ray_index] = distance_to_light - 2.0f * EPSILON;

	ray_buffer_shadow.illumination_and_pixel_index[shadow_ray_index] = make_float4(
		illumination.x,
		illumination.y,
		illumination.z,
		__int_as_float(pixel_index)
	);
}

extern "C" __global__ void kernel_shade_diffuse(int bounce, int sample_index) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.diffuse[bounce]) return;

	float3 ray_direction = ray_buffer_shade_diffuse_and_plastic.direction.get(index);
	RayHit hit           = ray_buffer_shade_diffuse_and_plastic.hits     .get(index);

	int ray_pixel_index = ray_buffer_shade_diffuse_and_plastic.pixel_index[index];
	int x = ray_pixel_index % screen_pitch;
	int y = ray_pixel_index / screen_pitch;

	float3 ray_throughput;
	if (bounce == 0) {
		ray_throughput = make_float3(1.0f); // Throughput is known to be (1,1,1) still, skip the global memory load
	} else {
		ray_throughput = ray_buffer_shade_diffuse_and_plastic.throughput.get(index);
	}

	int material_id = mesh_get_material_id(hit.mesh_id);
	MaterialDiffuse material = material_as_diffuse(material_id);

	// Obtain hit Triangle position, normal, and texture coordinates
	TrianglePosNorTex hit_triangle = triangle_get_positions_normals_and_tex_coords(hit.triangle_id);

	float3 hit_point;
	float3 hit_normal;
	float2 hit_tex_coord;
	triangle_barycentric(hit_triangle, hit.u, hit.v, hit_point, hit_normal, hit_tex_coord);

	float3 hit_point_local = hit_point; // Keep copy of the untransformed hit point in local space

	// Transform into world space
	Matrix3x4 world = mesh_get_transform(hit.mesh_id);
	matrix3x4_transform_position (world, hit_point);
	matrix3x4_transform_direction(world, hit_normal);

	hit_normal = normalize(hit_normal);
	if (dot(ray_direction, hit_normal) > 0.0f) hit_normal = -hit_normal;

	// Sample albedo
	float cone_angle;
	float cone_width;
	float3 albedo;
	if (config.enable_mipmapping) {
		albedo = sample_albedo(
			bounce,
			material.diffuse,
			material.texture_id,
			hit,
			hit_triangle,
			hit_point_local,
			hit_normal,
			hit_tex_coord,
			ray_direction,
			ray_buffer_shade_diffuse_and_plastic.cone,
			index,
			cone_angle, cone_width
		);
	} else {
		albedo = material_get_albedo(material.diffuse, material.texture_id, hit_tex_coord.x, hit_tex_coord.y);
	}

	if (bounce > 0) {
		ray_throughput *= albedo;
	} else if (config.enable_albedo || config.enable_svgf) {
		frame_buffer_albedo[ray_pixel_index] = make_float4(albedo);
	}

	if (bounce == 0 && config.enable_svgf) {
		float3 hit_point_prev = hit_point_local;

		Matrix3x4 world_prev = mesh_get_transform_prev(hit.mesh_id);
		matrix3x4_transform_position(world_prev, hit_point_prev);

		svgf_set_gbuffers(x, y, hit, hit_point, hit_normal, hit_point_prev);
	}

	if (config.enable_next_event_estimation && lights_total_weight > 0.0f) {
		nee_sample(ray_pixel_index, bounce, sample_index, hit_point, hit_normal, ray_throughput, [&](const float3 & to_light, float cos_theta, float3 & brdf, float & pdf) {
			if (cos_theta <= 0.0f) return false;

			brdf = make_float3(cos_theta * ONE_OVER_PI);
			pdf  = cos_theta * ONE_OVER_PI;

			return pdf_is_valid(pdf);
		});
	}

	float3 hit_tangent, hit_binormal; orthonormal_basis(hit_normal, hit_tangent, hit_binormal);

	float2 rand_brdf = random<SampleDimension::BRDF>(ray_pixel_index, bounce, sample_index);
	float3 direction_local = sample_cosine_weighted_direction(rand_brdf.x, rand_brdf.y);

	float3 direction_out = local_to_world(direction_local, hit_tangent, hit_binormal, hit_normal);
	float3 origin_out    = ray_origin_epsilon_offset(hit_point, direction_out, hit_normal);

	int index_out = atomic_agg_inc(&buffer_sizes.trace[bounce + 1]);

	ray_buffer_trace.origin   .set(index_out, origin_out);
	ray_buffer_trace.direction.set(index_out, direction_out);

	if (config.enable_mipmapping) {
		ray_buffer_trace.cone[index_out] = make_float2(cone_angle, cone_width);
	}

	ray_buffer_trace.pixel_index_and_mis_eligable[index_out] = ray_pixel_index | (true << 31);
	ray_buffer_trace.throughput.set(index_out, ray_throughput);

	ray_buffer_trace.last_pdf[index_out] = fabsf(dot(direction_out, hit_normal)) * ONE_OVER_PI;
}

extern "C" __global__ void kernel_shade_plastic(int bounce, int sample_index) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.plastic[bounce]) return;

	index = (BATCH_SIZE - 1) - index;

	float3 ray_direction = ray_buffer_shade_diffuse_and_plastic.direction.get(index);
	RayHit hit           = ray_buffer_shade_diffuse_and_plastic.hits     .get(index);

	int ray_pixel_index = ray_buffer_shade_diffuse_and_plastic.pixel_index[index];
	int x = ray_pixel_index % screen_pitch;
	int y = ray_pixel_index / screen_pitch;

	float3 ray_throughput;
	if (bounce == 0) {
		ray_throughput = make_float3(1.0f); // Throughput is known to be (1,1,1) still, skip the global memory load
	} else {
		ray_throughput = ray_buffer_shade_diffuse_and_plastic.throughput.get(index);
	}

	int material_id = mesh_get_material_id(hit.mesh_id);
	MaterialPlastic material = material_as_plastic(material_id);

	// Obtain hit Triangle position, normal, and texture coordinates
	TrianglePosNorTex hit_triangle = triangle_get_positions_normals_and_tex_coords(hit.triangle_id);

	float3 hit_point;
	float3 hit_normal;
	float2 hit_tex_coord;
	triangle_barycentric(hit_triangle, hit.u, hit.v, hit_point, hit_normal, hit_tex_coord);

	float3 hit_point_local = hit_point; // Keep copy of the untransformed hit point in local space

	// Transform into world space
	Matrix3x4 world = mesh_get_transform(hit.mesh_id);
	matrix3x4_transform_position (world, hit_point);
	matrix3x4_transform_direction(world, hit_normal);

	hit_normal = normalize(hit_normal);
	if (dot(ray_direction, hit_normal) > 0.0f) hit_normal = -hit_normal;

	// Sample albedo
	float cone_angle;
	float cone_width;
	float3 albedo;
	if (config.enable_mipmapping) {
		albedo = sample_albedo(
			bounce,
			material.diffuse,
			material.texture_id,
			hit,
			hit_triangle,
			hit_point_local,
			hit_normal,
			hit_tex_coord,
			ray_direction,
			ray_buffer_shade_diffuse_and_plastic.cone,
			index,
			cone_angle, cone_width
		);
	} else {
		albedo = material_get_albedo(material.diffuse, material.texture_id, hit_tex_coord.x, hit_tex_coord.y);
	}

	if (bounce == 0 && (config.enable_albedo || config.enable_svgf)) {
		frame_buffer_albedo[ray_pixel_index] = make_float4(1.0f);
	}

	if (bounce == 0 && config.enable_svgf) {
		float3 hit_point_prev = hit_point_local;

		Matrix3x4 world_prev = mesh_get_transform_prev(hit.mesh_id);
		matrix3x4_transform_position(world_prev, hit_point_prev);

		svgf_set_gbuffers(x, y, hit, hit_point, hit_normal, hit_point_prev);
	}

	float3 hit_tangent, hit_binormal; orthonormal_basis(hit_normal, hit_tangent, hit_binormal);

	float3 omega_i = world_to_local(-ray_direction, hit_tangent, hit_binormal, hit_normal);

	float alpha_x = material.roughness;
	float alpha_y = material.roughness;
	
	constexpr float ETA = 1.0f / 1.5f;
	constexpr float TIR_COMPENSATION = 0.596345782f; // Hemispherical integral of fresnel * cos(theta)

	if (config.enable_next_event_estimation && lights_total_weight > 0.0f) {
		nee_sample(ray_pixel_index, bounce, sample_index, hit_point, hit_normal, ray_throughput, [&](const float3 & to_light, float cos_theta, float3 & brdf, float & pdf) {
			if (cos_theta <= 0.0f) return false;

			float3 omega_o = world_to_local(to_light, hit_tangent, hit_binormal, hit_normal);
			float3 omega_m = normalize(omega_i + omega_o);

			// Specular component
			float F  = fresnel_dielectric(dot(omega_i, omega_m), ETA);
			float D  = ggx_D (omega_m, alpha_x, alpha_y);
			float G1 = ggx_G1(omega_i, alpha_x, alpha_y);
			float G2 = ggx_G2(omega_o, omega_i, omega_m, alpha_x, alpha_y);

			float3 brdf_specular = make_float3(F * G2 * D / (4.0f * omega_i.z));

			// Diffuse component
			float F_i = fresnel_dielectric(omega_i.z, ETA);
			float F_o = fresnel_dielectric(omega_o.z, ETA);

			float3 brdf_diffuse = ETA*ETA * (1.0f - F_i) * (1.0f - F_o) * albedo * ONE_OVER_PI / (1.0f - albedo * TIR_COMPENSATION) * omega_o.z;

			float pdf_specular = G1 * D / (4.0f * omega_i.z);
			float pdf_diffuse  = omega_o.z * ONE_OVER_PI;

			pdf  = lerp(pdf_diffuse, pdf_specular, F_i);
			brdf = brdf_specular + brdf_diffuse; // BRDF * cos(theta_o)

			return pdf_is_valid(pdf);
		});
	}

	float  rand_fresnel = random<SampleDimension::RUSSIAN_ROULETTE>(ray_pixel_index, bounce, sample_index).y;
	float2 rand_brdf    = random<SampleDimension::BRDF>            (ray_pixel_index, bounce, sample_index);

	float F_i = fresnel_dielectric(omega_i.z, ETA);

	float3 omega_m;
	float3 omega_o;
	if (rand_fresnel < F_i) {
		// Sample specular component
		omega_m = sample_visible_normals_ggx(omega_i, material.roughness, material.roughness, rand_brdf.x, rand_brdf.y);
		omega_o = reflect(-omega_i, omega_m);
	} else {
		// Sample diffuse component
		omega_o = sample_cosine_weighted_direction(rand_brdf.x, rand_brdf.y);
		omega_m = normalize(omega_i + omega_o);
	}

	if (omega_m.z < 0.0f) return; // Wrong hemisphere

	// Specular component
	float F  = fresnel_dielectric(dot(omega_i, omega_m), ETA);
	float D  = ggx_D (omega_m, alpha_x, alpha_y);
	float G1 = ggx_G1(omega_i, alpha_x, alpha_y);
	float G2 = ggx_G2(omega_o, omega_i, omega_m, alpha_x, alpha_y);

	float3 brdf_specular = make_float3(F * G2 * D / (4.0f * omega_i.z));

	// Diffuse component
	float F_o = fresnel_dielectric(omega_o.z, ETA);

	float3 brdf_diffuse = ETA*ETA * (1.0f - F_i) * (1.0f - F_o) * albedo * ONE_OVER_PI / (1.0f - albedo * TIR_COMPENSATION) * omega_o.z;

	// PDFs
	float pdf_specular = G1 * D / (4.0f * omega_i.z);
	float pdf_diffuse  = omega_o.z * ONE_OVER_PI;
	float pdf          = lerp(pdf_diffuse, pdf_specular, F_i);

	if (!pdf_is_valid(pdf)) return;

	ray_throughput *= (brdf_specular + brdf_diffuse) / pdf; // BRDF * cos(theta) / pdf

	float3 direction_out = local_to_world(omega_o, hit_tangent, hit_binormal, hit_normal);
	float3 origin_out = ray_origin_epsilon_offset(hit_point, direction_out, hit_normal);

	int index_out = atomic_agg_inc(&buffer_sizes.trace[bounce + 1]);

	ray_buffer_trace.origin   .set(index_out, origin_out);
	ray_buffer_trace.direction.set(index_out, direction_out);

	if (config.enable_mipmapping) {
		ray_buffer_trace.cone[index_out] = make_float2(cone_angle, cone_width);
	}

	ray_buffer_trace.pixel_index_and_mis_eligable[index_out] = ray_pixel_index | (true << 31);
	ray_buffer_trace.throughput.set(index_out, ray_throughput);

	ray_buffer_trace.last_pdf[index_out] = pdf;
}

extern "C" __global__ void kernel_shade_dielectric(int bounce, int sample_index) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.dielectric[bounce]) return;

	float3 ray_direction = ray_buffer_shade_dielectric_and_conductor.direction.get(index);
	RayHit hit           = ray_buffer_shade_dielectric_and_conductor.hits     .get(index);

	int ray_pixel_index = ray_buffer_shade_dielectric_and_conductor.pixel_index[index];

	float3 ray_throughput;
	if (bounce == 0) {
		ray_throughput = make_float3(1.0f); // Throughput is known to be (1,1,1) still, skip the global memory load
	} else {
		ray_throughput = ray_buffer_shade_dielectric_and_conductor.throughput.get(index);
	}

	ASSERT(hit.triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	int material_id = mesh_get_material_id(hit.mesh_id);
	MaterialDielectric material = material_as_dielectric(material_id);

	// Obtain hit Triangle position, normal, and texture coordinates
	TrianglePosNor hit_triangle = triangle_get_positions_and_normals(hit.triangle_id);

	float3 hit_point;
	float3 hit_normal;
	triangle_barycentric(hit_triangle, hit.u, hit.v, hit_point, hit_normal);

	// Transform into world space
	Matrix3x4 world = mesh_get_transform(hit.mesh_id);
	matrix3x4_transform_position (world, hit_point);
	matrix3x4_transform_direction(world, hit_normal);

	hit_normal = normalize(hit_normal);

	bool entering_material = dot(ray_direction, hit_normal) < 0.0f;
	if (!entering_material) {
		hit_normal = -hit_normal;

		// Lambert-Beer Law
		// NOTE: does not take into account e.g. nested dielectrics or diffuse inside dielectric!
		ray_throughput.x *= expf(material.negative_absorption.x * hit.t);
		ray_throughput.y *= expf(material.negative_absorption.y * hit.t);
		ray_throughput.z *= expf(material.negative_absorption.z * hit.t);
	}

	// Construct orthonormal basis
	float3 hit_tangent, hit_binormal;
	orthonormal_basis(hit_normal, hit_tangent, hit_binormal);

	float3 omega_i = world_to_local(-ray_direction, hit_tangent, hit_binormal, hit_normal);

	float eta = entering_material ? 1.0f / material.index_of_refraction : material.index_of_refraction;

	float alpha_x = material.roughness;
	float alpha_y = material.roughness;
	
	if (config.enable_next_event_estimation && lights_total_weight > 0.0f && material.roughness >= ROUGHNESS_CUTOFF) {
		nee_sample(ray_pixel_index, bounce, sample_index, hit_point, hit_normal, ray_throughput, [&](const float3 & to_light, float cos_theta, float3 & brdf, float & pdf) {
			float3 omega_o = world_to_local(to_light, hit_tangent, hit_binormal, hit_normal);

			bool reflected = omega_o.z >= 0.0f; // Same sign means reflection, alternate signs means transmission

			float3 omega_m;
			if (reflected) {
				omega_m = normalize(omega_i + omega_o);
			} else {
				omega_m = normalize(eta * omega_i + omega_o);
			}
			omega_m *= sign(omega_m.z);

			float i_dot_m = abs_dot(omega_i, omega_m);
			float o_dot_m = abs_dot(omega_o, omega_m);

			float F  = fresnel_dielectric(i_dot_m, eta);
			float D  = ggx_D (omega_m, alpha_x, alpha_y);
			float G1 = ggx_G1(omega_i, alpha_x, alpha_y);
			float G2 = ggx_G2(omega_o, omega_i, omega_m, alpha_x, alpha_y);

			if (reflected) {
				pdf = F * G1 * D / (4.0f * omega_i.z);

				brdf = make_float3(F * G2 * D / (4.0f * omega_i.z)); // BRDF times cos(theta_o)
			} else {
				if (F >= 0.999f) return false; // TIR, no transmission possible

				pdf = (1.0f - F) * G1 * D * i_dot_m * o_dot_m / (omega_i.z * square(eta * i_dot_m + o_dot_m));

				brdf = eta * eta * make_float3((1.0f - F) * G2 * D * i_dot_m * o_dot_m / (omega_i.z * square(eta * i_dot_m + o_dot_m))); // BRDF times cos(theta_o)
			}

			return pdf_is_valid(pdf);
		});
	}

	float  rand_fresnel = random<SampleDimension::RUSSIAN_ROULETTE>(ray_pixel_index, bounce, sample_index).y;
	float2 rand_brdf    = random<SampleDimension::BRDF>            (ray_pixel_index, bounce, sample_index);

	float3 omega_m = sample_visible_normals_ggx(omega_i, material.roughness, material.roughness, rand_brdf.x, rand_brdf.y);

	float F = fresnel_dielectric(abs_dot(omega_i, omega_m), eta);
	bool reflected = rand_fresnel < F;

	float3 omega_o;
	if (reflected) {
		omega_o = 2.0f * dot(omega_i, omega_m) * omega_m - omega_i;
	} else {
		float k = 1.0f - eta*eta * (1.0f - square(dot(omega_i, omega_m)));
		omega_o = (eta * abs_dot(omega_i, omega_m) - sqrtf(k)) * omega_m - eta * omega_i;
	}

	if (reflected ^ (omega_o.z >= 0.0f)) return; // Hemisphere check: reflection should have positive z, transmission negative z

	float D  = ggx_D (omega_m, alpha_x, alpha_y);
	float G1 = ggx_G1(omega_i, alpha_x, alpha_y);
	float G2 = ggx_G2(omega_o, omega_i, omega_m, alpha_x, alpha_y);

	float i_dot_m = abs_dot(omega_i, omega_m);
	float o_dot_m = abs_dot(omega_o, omega_m);

	float pdf;
	if (reflected) {
		pdf = F * G1 * D / (4.0f * omega_i.z);
	} else {
		pdf = (1.0f - F) * G1 * D * i_dot_m * o_dot_m / (omega_i.z * square(eta * i_dot_m + o_dot_m));

		ray_throughput *= eta*eta; // Account for solid angle compression
	}

	ray_throughput *= G2 / G1; // BRDF * cos(theta_o) / pdf (same for reflection and transmission)

	float3 direction_out = local_to_world(omega_o, hit_tangent, hit_binormal, hit_normal);
	float3 origin_out    = ray_origin_epsilon_offset(hit_point, direction_out, hit_normal);

	if (bounce == 0 && (config.enable_albedo || config.enable_svgf)) {
		frame_buffer_albedo[ray_pixel_index] = make_float4(1.0f);
	}

	int index_out = atomic_agg_inc(&buffer_sizes.trace[bounce + 1]);

	ray_buffer_trace.origin   .set(index_out, origin_out);
	ray_buffer_trace.direction.set(index_out, direction_out);

	if (config.enable_mipmapping) {
		float cone_angle;
		float cone_width;
		if (bounce == 0) {
			cone_angle = camera.pixel_spread_angle;
			cone_width = 0.0f;
		} else {
			float2 cone = ray_buffer_shade_dielectric_and_conductor.cone[index];
			cone_angle = cone.x;
			cone_width = cone.y;
		}

		float mesh_scale = mesh_get_scale(hit.mesh_id);

		float curvature = triangle_get_curvature(
			hit_triangle.position_edge_1,
			hit_triangle.position_edge_2,
			hit_triangle.normal_edge_1,
			hit_triangle.normal_edge_2
		) / mesh_scale;

		cone_width += cone_angle * hit.t;
		cone_angle += -2.0f * curvature * fabsf(cone_width) / dot(hit_normal, ray_direction); // Eq. 5 (Akenine-Möller 2021)

		ray_buffer_trace.cone[index_out] = make_float2(cone_angle, cone_width);
	}

	ray_buffer_trace.pixel_index_and_mis_eligable[index_out] = ray_pixel_index | ((material.roughness >= ROUGHNESS_CUTOFF) << 31);
	ray_buffer_trace.throughput.set(index_out, ray_throughput);

	ray_buffer_trace.last_pdf[index_out] = pdf;
}

extern "C" __global__ void kernel_shade_conductor(int bounce, int sample_index) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= buffer_sizes.conductor[bounce]) return;

	index = (BATCH_SIZE - 1) - index;

	float3 ray_direction = ray_buffer_shade_dielectric_and_conductor.direction.get(index);

	RayHit hit = ray_buffer_shade_dielectric_and_conductor.hits.get(index);

	int ray_pixel_index = ray_buffer_shade_dielectric_and_conductor.pixel_index[index];
	int x = ray_pixel_index % screen_pitch;
	int y = ray_pixel_index / screen_pitch;

	float3 ray_throughput;
	if (bounce == 0) {
		ray_throughput = make_float3(1.0f);	// Throughput is known to be (1,1,1) still, skip the global memory load
	} else {
		ray_throughput = ray_buffer_shade_dielectric_and_conductor.throughput.get(index);
	}

	ASSERT(hit.triangle_id != -1, "Ray must have hit something for this Kernel to be invoked!");

	int material_id = mesh_get_material_id(hit.mesh_id);
	MaterialConductor material = material_as_conductor(material_id);

	// Obtain hit Triangle position, normal, and texture coordinates
	TrianglePosNorTex hit_triangle = triangle_get_positions_normals_and_tex_coords(hit.triangle_id);

	float3 hit_point;
	float3 hit_normal;
	float2 hit_tex_coord;
	triangle_barycentric(hit_triangle, hit.u, hit.v, hit_point, hit_normal, hit_tex_coord);

	float3 hit_point_local = hit_point; // Keep copy of the untransformed hit point in local space

	// Transform into world space
	Matrix3x4 world = mesh_get_transform(hit.mesh_id);
	matrix3x4_transform_position (world, hit_point);
	matrix3x4_transform_direction(world, hit_normal);

	hit_normal = normalize(hit_normal);
	if (dot(ray_direction, hit_normal) > 0.0f) hit_normal = -hit_normal;

	// Sample albedo
	float cone_angle;
	float cone_width;
	float3 albedo;
	if (config.enable_mipmapping) {
		albedo = sample_albedo(
			bounce,
			material.diffuse,
			material.texture_id,
			hit,
			hit_triangle,
			hit_point_local,
			hit_normal,
			hit_tex_coord,
			ray_direction,
			ray_buffer_shade_dielectric_and_conductor.cone,
			index,
			cone_angle, cone_width
		);
	} else {
		albedo = material_get_albedo(material.diffuse, material.texture_id, hit_tex_coord.x, hit_tex_coord.y);
	}

	if (bounce > 0) {
		ray_throughput *= albedo;
	} else if (config.enable_albedo || config.enable_svgf) {
		frame_buffer_albedo[ray_pixel_index] = make_float4(albedo);
	}

	if (bounce == 0 && config.enable_svgf) {
		float3 hit_point_prev = hit_point_local;

		Matrix3x4 world_prev = mesh_get_transform_prev(hit.mesh_id);
		matrix3x4_transform_position(world_prev, hit_point_prev);

		svgf_set_gbuffers(x, y, hit, hit_point, hit_normal, hit_point_prev);
	}

	// Construct orthonormal basis
	float3 hit_tangent, hit_binormal;
	orthonormal_basis(hit_normal, hit_tangent, hit_binormal);

	float3 omega_i = world_to_local(-ray_direction, hit_tangent, hit_binormal, hit_normal);
	
	float alpha_x = material.roughness;
	float alpha_y = material.roughness; // TODO: anisotropic
	
	if (config.enable_next_event_estimation && lights_total_weight > 0.0f && material.roughness >= ROUGHNESS_CUTOFF) {
		nee_sample(ray_pixel_index, bounce, sample_index, hit_point, hit_normal, ray_throughput, [&](const float3 & to_light, float cos_theta, float3 & brdf, float & pdf) {
			if (cos_theta <= 0.0f) return false;

			float3 omega_o = world_to_local(to_light, hit_tangent, hit_binormal, hit_normal);
			float3 omega_m = normalize(omega_o + omega_i);
			
			float mu = dot(omega_o, omega_m);
			if (mu <= 0.0f) return false;

			float3 F  = fresnel_conductor(mu, material.eta, material.k);
			float  D  = ggx_D (omega_m, alpha_x, alpha_y);
			float  G1 = ggx_G1(omega_i, alpha_x, alpha_y);
			float  G2 = ggx_G2(omega_o, omega_i, omega_m, alpha_x, alpha_y);

			pdf  =     G1 * D / (4.0f * omega_i.z);
			brdf = F * G2 * D / (4.0f * omega_i.z); // BRDF * cos(theta_o)

			return pdf_is_valid(pdf);
		});
	}

	// Importance sample distribution of normals
	float2 rand_brdf = random<SampleDimension::BRDF>(ray_pixel_index, bounce, sample_index);

	float3 omega_m = sample_visible_normals_ggx(omega_i, alpha_x, alpha_y, rand_brdf.x, rand_brdf.y);
	float3 omega_o = reflect(-omega_i, omega_m);

	float o_dot_m = dot(omega_o, omega_m);
	if (o_dot_m <= 0.0f) return;

	float3 F  = fresnel_conductor(o_dot_m, material.eta, material.k);
	float  D  = ggx_D (omega_m, alpha_x, alpha_y);
	float  G1 = ggx_G1(omega_i, alpha_x, alpha_y);
	float  G2 = ggx_G2(omega_o, omega_i, omega_m, alpha_x, alpha_y);

	float pdf = G1 * D / (4.0f * omega_i.z);

	ray_throughput *= F * G2 / G1; // BRDF * cos(theta_o) / pdf

	float3 direction_out = local_to_world(omega_o, hit_tangent, hit_binormal, hit_normal);
	float3 origin_out    = ray_origin_epsilon_offset(hit_point, direction_out, hit_normal);

	int index_out = atomic_agg_inc(&buffer_sizes.trace[bounce + 1]);

	ray_buffer_trace.origin   .set(index_out, origin_out);
	ray_buffer_trace.direction.set(index_out, direction_out);

	if (config.enable_mipmapping) {
		ray_buffer_trace.cone[index_out] = make_float2(cone_angle, cone_width);
	}

	ray_buffer_trace.pixel_index_and_mis_eligable[index_out] = ray_pixel_index | ((material.roughness >= ROUGHNESS_CUTOFF) << 31);
	ray_buffer_trace.throughput.set(index_out, ray_throughput);

	ray_buffer_trace.last_pdf[index_out] = pdf;
}

extern "C" __global__ void kernel_accumulate(float frames_accumulated) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= screen_width || y >= screen_height) return;

	int pixel_index = x + y * screen_pitch;

	float4 direct   = frame_buffer_direct  [pixel_index];
	float4 indirect = frame_buffer_indirect[pixel_index];

	float4 colour = direct + indirect;

	if (config.enable_albedo) {
		colour *= frame_buffer_albedo[pixel_index];
	}

	if (frames_accumulated > 0.0f) {
		float4 colour_prev = accumulator.get(x, y);

		colour = colour_prev + (colour - colour_prev) / frames_accumulated; // Online average
	}

//	if (isnan(colour.x + colour.y + colour.z)) colour = make_float4(1,0,1,1);

	accumulator.set(x, y, colour);
}
